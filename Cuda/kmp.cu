#include "hip/hip_runtime.h"
% % cu
#include <bits/stdc++.h>
	using namespace std;
#define M 1024

void computeLPS(string pat, int m, int lps[])
{
	int len = 0, i = 1;
	lps[0] = 0;
	while (i < m)
	{
		if (pat[i] == pat[len])
		{
			lps[i] = len + 1;
			len++;
			i++;
		}
		else
		{
			if (len != 0)
				len = lps[len - 1];
			else
			{
				lps[i] = 0;
				i++;
			}
		}
	}
}

__global__ void kmpSearch(char *txt, char *pat, int n, int m, int lps[], int res[])
{
	int pid = blockIdx.x * blockDim.x + threadIdx.x;
	int cp;
	int start = pid, end = pid + m;
	int t = end - start + 1;
	if (pid < n - m)
	{
		int i = pid, flag = 0, j = 0;
		while (i < start + t - 1)
		{
			if (txt[i] == pat[j])
			{
				i++;
				j++;
			}
			else
			{
				if (j != 0)
					cp = lps[j - 1];
				i++;
			}
			if (j == m)
			{
				flag = 1;
				cp = lps[j - 1];
			}
		}
		res[pid] = flag;
	}
}

int main()
{
	int blocks[] = {1, 1, 1, 1, 1, 1, 1, 10, 20, 30, 40, 50, M / 8, M / 4, M / 2, M, M, M, M, M};
	int threads[] = {1, 10, 20, 30, 40, 50, M, 10, 10, 10, 10, 10, M, M, M, M / 8, M / 4, M / 2, M};

	//Randomly generating DNA Sequence;
	vector<char> list = {'a', 'g', 'c', 't'};
	int n = 1000000; //Enter number of characters
	cout << "\nDNA Sequence is Randomly Generated\n";
	srand(time(0));
	{
		ofstream cout("dna.txt");
		for (long i = 0; i < n; i++)
		{
			int random = rand() % 4;
			cout << list[random];
		}
	}
	char txt[n + 1];
	char *d_txt, *d_pat; //device copies
	int *d_lps, *d_res;
	string s, p = "agctac"; //Enter the pattern

	//reading dna sequence from file
	{
		ifstream cin("dna.txt");
		cin >> s;
	}
	strcpy(txt, s.c_str());

	int m = p.size();
	char pat[m + 1];
	strcpy(pat, p.c_str());

	cout << endl;
	cout << "DNA Sequence : " << s << endl;
	cout << "Pattern      : " << p << endl;

	int *lps;
	int res[n] = {0};
	lps = new int[m]; //longest proper prefix suffix
	computeLPS(pat, m, lps);
	cout << "\nLength of DNA Sequence : " << n << endl;
	cout << "Length of Pattern      : " << m << endl
		 << endl;

	// Allocate space for device copies of txt, pat,lps and res
	hipMalloc((void **)&d_txt, n * sizeof(char));
	hipMalloc((void **)&d_pat, m * sizeof(char));
	hipMalloc((void **)&d_lps, m * sizeof(int));
	hipMalloc((void **)&d_res, n * sizeof(int));

	hipMemcpy(d_txt, &txt, n * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_pat, &pat, m * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_lps, &lps, m * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_res, &res, n * sizeof(int), hipMemcpyHostToDevice);

	for (int k = 0; k < 19; k++)
	{
		float elapsed = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start, 0);
		kmpSearch<<<blocks[k], threads[k]>>>(d_txt, d_pat, n, m, d_lps, d_res);

		// Copy result back to host
		hipError_t err = hipMemcpy(&res, d_res, n * sizeof(int), hipMemcpyDeviceToHost);
		if (err != hipSuccess)
			cout << "CUDA Error copying to Host :" << hipGetErrorString(err) << endl;

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&elapsed, start, stop);

		hipEventDestroy(start);
		hipEventDestroy(stop);

		printf("Blocks = %4d and Threads per Block = %4d Time = %.5f\n", blocks[k], threads[k], elapsed);
	}

	// Cleanup
	hipFree(d_txt);
	hipFree(d_pat);
	hipFree(d_lps);
	hipFree(d_res);
	int cnt = 0;
	cout << "\n\n*********** Final Results ***********\n";
	for (int i = 0; i < n; i++)
		if (res[i] != 0)
			cnt++;
	cout << "\nTotal number of matches = " << cnt << endl
		 << endl;
	int counter = 0;
	for (int i = 0; i < n; i++)
	{
		if (res[i] != 0)
		{
			counter++;
			cout << "Match - " << counter << " to '" << pat << "' found at : " << i << " - " << i + m - 1 << endl;
		}
	}

	return 0;
}